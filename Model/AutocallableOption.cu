#include "hip/hip_runtime.h"
//
// Created by marco on 05/05/22.
//

#include <iostream>

#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <random>

#include "AutocallableOption.cuh"

__global__ void K_call_payoff(AutoCallableOption option, float *d_samples, float *d_normals, int n_paths,
                              float *d_observationDates, float *d_barriers, float *d_payoffs, int dateBarrierSize){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i >= n_paths) return;

    float barrier_hit = false;
    float S = option.asset.spot_price;
    int date_index = 0;
    float dt = d_observationDates[date_index];

    while (date_index <= dateBarrierSize - 1) {
        S = S * exp((option.asset.risk_free_rate - (pow(option.asset.volatility, 2) / 2)) * dt
                + option.asset.volatility * sqrt(dt) * d_normals[i + date_index]);

        if (S >= d_barriers[date_index]) { barrier_hit = true; break; }

        date_index++;
        dt = d_observationDates[date_index] - d_observationDates[date_index - 1];
    }

    if(!barrier_hit)
        d_samples[i] = exp(-option.asset.risk_free_rate * d_observationDates[dateBarrierSize-1]) * option.rebase;
    else
        d_samples[i] = exp(-option.asset.risk_free_rate * d_observationDates[date_index]) * d_payoffs[date_index];
}

AutoCallableOption::AutoCallableOption(){} // Definire meglio questo costruttore

AutoCallableOption::AutoCallableOption(Asset asset, float rebase, vector<float> observationDates,
                                       vector<float> barriers,  vector<float> payoffs){
    this->asset = asset;
    this->observationDates = observationDates;
    this->barriers = barriers;
    this->payoffs = payoffs;
    this->rebase = rebase;
}

AutoCallableOption::AutoCallableOption(AutoCallableOption &option){
    this->asset = option.asset;
    this->n_intervals = 3 * 365; // must take the last observation date
    this->dt = 1.0/(float) 365; // day increment
    this->rebase = option.rebase;
    this->dateBarrier = option.dateBarrier;
}

SimulationResult AutoCallableOption::call_payoff_montecarlo_cpu(){
    mt19937 gen(static_cast<long unsigned int>(time(0)));
    normal_distribution<double> distribution(0.0f, 1.0f);

    const int N_PATHS = 10000000;
    bool barrier_hit = false;

    float S = asset.spot_price;
    float z = 0;

    float *C = (float *)malloc(sizeof(float) * N_PATHS);

    int date_index = 0;
    float dt = 0.0f;

    for(int i=0; i<N_PATHS; i++) {
        barrier_hit = false;
        S = asset.spot_price;
        date_index = 0;
        dt = observationDates[date_index];
        while (date_index <= observationDates.size() - 1) {
            z = distribution(gen);
            S = S * exp((asset.risk_free_rate - (pow(asset.volatility, 2) / 2)) * dt + asset.volatility * sqrt(dt) * z);

            if (S >= barriers[date_index]) {
                    barrier_hit = true;
                    break;
            }

            date_index++;
            dt = observationDates[date_index] - observationDates[date_index - 1];
        }

        if(!barrier_hit) C[i] = exp(-asset.risk_free_rate * observationDates[observationDates.size()-1]) * rebase;
        else C[i] = exp(-asset.risk_free_rate * observationDates[date_index]) * payoffs[date_index];
    }

    SimulationResult result(C, N_PATHS, 0);
    return result;
}

SimulationResult  AutoCallableOption::call_payoff_montecarlo_gpu(){
    const int N_THREADS = 256;
    const int N_PATHS = 10000000;

    size_t size = sizeof(float) * N_PATHS;
    float *h_samples = (float *) malloc(size);

    float *d_samples = nullptr;
    hipMalloc((void **)&d_samples, size);

    float *d_normals = nullptr;
    hipMalloc((void **)&d_normals, size * observationDates.size());

    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL);
    hiprandGenerateNormal(generator, d_normals, N_PATHS * observationDates.size(), 0.0f, 1.0f);
    //hipMemcpy(h_normals, d_normals, size, hipMemcpyDeviceToHost);

    size_t observationSize = sizeof(float) * observationDates.size();

    float *d_observationDates;
    hipMalloc((void**)&d_observationDates, observationSize);
    hipMemcpy(d_observationDates, observationDates.data(), observationSize, hipMemcpyHostToDevice);

    float *d_barriers;
    hipMalloc((void**)&d_barriers, observationSize);
    hipMemcpy(d_barriers, barriers.data(), observationSize, hipMemcpyHostToDevice);

    float *d_payoffs;
    hipMalloc((void**)&d_payoffs, observationSize);
    hipMemcpy(d_payoffs, payoffs.data(), observationSize, hipMemcpyHostToDevice);

    AutoCallableOption option(*this);
    K_call_payoff<<<(N_PATHS + N_THREADS - 1)/N_THREADS, N_THREADS>>>(option, d_samples, d_normals, N_PATHS, d_observationDates, d_barriers, d_payoffs, observationDates.size());

    hipDeviceSynchronize();

    // copy results from device to host
    hipMemcpy(h_samples, d_samples, size, hipMemcpyDeviceToHost);

    hiprandDestroyGenerator(generator);

    SimulationResult result(h_samples, N_PATHS, 0);

    free(h_samples);

    hipFree(d_samples);
    hipFree(d_normals);
    hipFree(d_observationDates);
    hipFree(d_samples);
    hipFree(d_barriers);

    return result;
}