//
// Created by marco on 21/06/22.
//

#include "EuropeanOptionCPU.cuh"
#include "../../Shared/SharedFunctions.cuh"
#include "../../../StatisticUtils/StatisticsSerialCPU.cuh"
#include "../../../StatisticUtils/StatisticsCPU.cuh"

#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <omp.h>
#include <iostream>

EuropeanOptionCPU::EuropeanOptionCPU(Asset *asset, float strikePrice, float timeToMaturity,
                                     MonteCarloParams *monteCarloParams) : EuropeanOption(asset, strikePrice,
                                                                                          timeToMaturity),
                                                                           monteCarloParams(monteCarloParams) {}

MonteCarloParams *EuropeanOptionCPU::getMonteCarloParams() const {
    return monteCarloParams;
}

void EuropeanOptionCPU::setMonteCarloParams(MonteCarloParams *monteCarloParams) {
    EuropeanOptionCPU::monteCarloParams = monteCarloParams;
}

SimulationResult EuropeanOptionCPU::callPayoff() {
    const int N_SIMULATIONS = getMonteCarloParams()->getNSimulations();
    float *samples = (float *)malloc(N_SIMULATIONS * sizeof (float));

    size_t size = sizeof(float) * N_SIMULATIONS;
    float *h_normals = (float *) malloc(size);
    float *d_normals = nullptr;
    hipMalloc((void **)&d_normals, size);

    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, monteCarloParams->getRngType());
    hiprandSetPseudoRandomGeneratorSeed(generator, monteCarloParams->getSeed());
    hiprandGenerateNormal(generator, d_normals, N_SIMULATIONS, 0.0f, 1.0f);
    hipMemcpy(h_normals, d_normals, size, hipMemcpyDeviceToHost);

    // Start timer
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    #pragma omp parallel default(shared)
    {
        float S_T;
        #pragma omp for
        for(int i=0; i<N_SIMULATIONS; i++){
            S_T = generateS_T(getAsset()->getSpotPrice(), getAsset()->getRiskFreeRate(),
                              getAsset()->getVolatility(), timeToMaturity, h_normals[i]);
            samples[i] = discountCall(getAsset()->getRiskFreeRate(), timeToMaturity, S_T, strikePrice);
        }
    }

    StatisticsCPU statistics(samples, N_SIMULATIONS);
    statistics.calcMean();
    statistics.calcCI();

    // Calculate elapsed time
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    float elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()/1000000.0f;

    SimulationResult result(statistics.getMean(),
                            statistics.getConfidence(),
                            statistics.getStdError(),
                            elapsedTime);

    free(h_normals); hipFree(d_normals);
    free(samples);
    hiprandDestroyGenerator(generator);

    return result;
}

SimulationResult EuropeanOptionCPU::putPayoff() {
    const int N_SIMULATIONS = getMonteCarloParams()->getNSimulations();
    float *samples = (float *)malloc(N_SIMULATIONS * sizeof (float));

    size_t size = sizeof(float) * N_SIMULATIONS;
    float *h_normals = (float *) malloc(size);
    float *d_normals = nullptr;
    hipMalloc((void **)&d_normals, size);

    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, monteCarloParams->getRngType());
    hiprandSetPseudoRandomGeneratorSeed(generator, monteCarloParams->getSeed());
    hiprandGenerateNormal(generator, d_normals, N_SIMULATIONS, 0.0f, 1.0f);
    hipMemcpy(h_normals, d_normals, size, hipMemcpyDeviceToHost);

    // Start timer
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    #pragma omp parallel default(shared)
    {
        float S_T;
        #pragma omp for
        for(int i=0; i<N_SIMULATIONS; i++){
            S_T = generateS_T(getAsset()->getSpotPrice(), getAsset()->getRiskFreeRate(),
                              getAsset()->getVolatility(), timeToMaturity, h_normals[i]);
            samples[i] = discountPut(getAsset()->getRiskFreeRate(), timeToMaturity, S_T, strikePrice);
        }
    }

    StatisticsCPU statistics(samples, N_SIMULATIONS);
    statistics.calcMean();
    statistics.calcCI();

    // Calculate elapsed time
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    float elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()/1000000.0f;

    SimulationResult result(statistics.getMean(),
                            statistics.getConfidence(),
                            statistics.getStdError(),
                            elapsedTime);

    free(h_normals); hipFree(d_normals);
    free(samples);
    hiprandDestroyGenerator(generator);

    return result;
}

