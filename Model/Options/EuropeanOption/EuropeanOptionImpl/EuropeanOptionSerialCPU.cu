//
// Created by marco on 23/05/22.
//


#include "EuropeanOptionSerialCPU.cuh"
#include "../Shared/SharedFunctions.cuh"

#include "../../../Utilities/StatisticUtils/StatisticUtilsCPU.cuh"

#include <cmath>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>

EuropeanOptionSerialCPU::EuropeanOptionSerialCPU(Asset *asset, float strikePrice, float timeToMaturity,
                                                 MonteCarloParams *monteCarloParams)
        : EuropeanOption(asset, strikePrice, timeToMaturity), monteCarloParams(monteCarloParams) {}

MonteCarloParams *EuropeanOptionSerialCPU::getMonteCarloParams() const {
    return monteCarloParams;
}

void EuropeanOptionSerialCPU::setMonteCarloParams(MonteCarloParams *monteCarloParams) {
    EuropeanOptionSerialCPU::monteCarloParams = monteCarloParams;
}

SimulationResult EuropeanOptionSerialCPU::callPayoff() {
    const unsigned int N_SIMULATIONS = getMonteCarloParams()->getNSimulations();
    float *samples = (float *)malloc(N_SIMULATIONS * sizeof (float));

    size_t size = sizeof(float) * N_SIMULATIONS;
    float *h_normals = (float *) malloc(size);
    float *d_normals = nullptr;
    hipMalloc((void **)&d_normals, size);

    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, 42ULL);
    hiprandGenerateNormal(generator, d_normals, N_SIMULATIONS, 0.0f, 1.0f);
    hipMemcpy(h_normals, d_normals, size, hipMemcpyDeviceToHost);

    // Start timer
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    float S_T;
    for(unsigned int i=0; i<N_SIMULATIONS;i++){
        S_T = generateS_T(getAsset()->getSpotPrice(), getAsset()->getRiskFreeRate(),
                          getAsset()->getVolatility(), timeToMaturity, h_normals[i]);
        samples[i] = discountCall(getAsset()->getRiskFreeRate(), timeToMaturity, S_T, strikePrice);
    }

    StatisticUtilsCPU statistics(samples, N_SIMULATIONS);
    statistics.calcMean();
    statistics.calcCI();

    // Calculate elapsed time
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    float elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()/1000000.0f;

    SimulationResult result(statistics.getMean(), statistics.getConfidence(),statistics.getStdError(), elapsedTime);

    free(h_normals); hipFree(d_normals);
    free(samples);

    return result;
}

SimulationResult EuropeanOptionSerialCPU::putPayoff() {
    const int N_SIMULATIONS = getMonteCarloParams()->getNSimulations();
    float *samples = (float *)malloc(N_SIMULATIONS * sizeof (float));

    size_t size = sizeof(float) * N_SIMULATIONS;
    float *h_normals = (float *) malloc(size);
    float *d_normals = nullptr;
    hipMalloc((void **)&d_normals, size);

    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, 42ULL);
    hiprandGenerateNormal(generator, d_normals, N_SIMULATIONS, 0.0f, 1.0f);
    hipMemcpy(h_normals, d_normals, size, hipMemcpyDeviceToHost);

    // Start timer
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    float S_T;
    for(int i=0; i<N_SIMULATIONS;i++){
        S_T = generateS_T(getAsset()->getSpotPrice(), getAsset()->getRiskFreeRate(),
                          getAsset()->getVolatility(), timeToMaturity, h_normals[i]);
        samples[i] = discountPut(getAsset()->getRiskFreeRate(), timeToMaturity, S_T, strikePrice);
    }

    StatisticUtilsCPU statistics(samples, N_SIMULATIONS);
    statistics.calcMean();
    statistics.calcCI();

    // Calculate elapsed time
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    float elapsedTime = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()/1000000.0f;

    SimulationResult result(statistics.getMean(),
                            statistics.getConfidence(),
                            statistics.getStdError(),
                            elapsedTime);

    free(h_normals); hipFree(d_normals);
    free(samples);

    return result;
}

